// CUDA Implementation of the 2D wave equation

#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <math.h>

__device__ float src(int t, float dt) {
  float freqMag = 1000000000;  // 10^9 = GHz

  return 45 * sin(2 * acosf(-1.0) * 10 * freqMag * dt * t);
}

__global__ void Update(float* UOld, float* UNew, float* U, float* Ca, int SizeX,
                       int SizeY, int isrc, int jsrc, int t, float dt) {
  int r = blockDim.x * blockIdx.x + threadIdx.x;

  int i = r % SizeX;
  int j = (r - i) / SizeX;  // verify this

  int Idx = SizeX * i + j;
  int IdxLeft = SizeX * (i - 1) + j;
  int IdxRight = SizeX * (i + 1) + j;
  int IdxUp = SizeX * i + (j + 1);
  int IdxDown = SizeX * i + (j - 1);

  // Make sure we're in the grid loop
  if (i > 0 && i < (SizeX - 1) && j > 0 && j < (SizeY - 1))
    UNew[Idx] = 2 * U[Idx] - UOld[Idx] +
                Ca[Idx] * (U[IdxRight] + U[IdxLeft] + U[IdxUp] + U[IdxDown] -
                           4 * U[Idx]);

  if (i == isrc && j == isrc) UNew[Idx] = UNew[Idx] + src(t, dt);

  return;
}

// Mur Boundary Condition Kernel
__global__ void ApplyBC(float* UOld, float* UNew, float* U, int SizeX,
                        int SizeY, float dt, float dx) {
  int r = blockDim.x * blockIdx.x + threadIdx.x;

  int i = r % SizeX;
  int j = (r - i) / SizeX;  // verify this

  float cc = 299792458;

  float ABC_C1 = (cc * dt - dx) / (cc * dt + dx);
  float ABC_C2 = 2 * dx / (cc * dt + dx);
  float ABC_C3 = (cc * dt) * (cc * dt) / (2 * dx * (cc * dt + dx));

  int Idx = SizeX * i + j;
  int IdxLeft = SizeX * (i - 1) + j;
  int IdxRight = SizeX * (i + 1) + j;
  int IdxUp = SizeX * i + (j + 1);
  int IdxDown = SizeX * i + (j - 1);

  /**************
   * i == 0
   *************/
  if (i == 0 && j > 0 && j < (SizeY - 1)) {
    UNew[Idx] = -1 * UOld[IdxRight] + ABC_C1 * (UNew[IdxRight] + UOld[Idx]) +
                ABC_C2 * (U[Idx] + U[IdxRight]) +
                ABC_C3 * (U[IdxUp] - 2 * U[Idx] + U[IdxDown] + U[IdxRight + 1] -
                          2 * U[IdxRight] + U[IdxRight - 1]);
  }

  /**************
   * i == SizeX
   *************/
  if (i == SizeX - 1 && j > 0 && j < (SizeY - 1)) {
    UNew[Idx] = -1 * UOld[IdxLeft] + ABC_C1 * (UNew[IdxLeft] + UNew[Idx]) +
                ABC_C2 * (U[Idx] + U[IdxLeft]) +
                ABC_C3 * (U[IdxUp] - 2 * U[Idx] + U[IdxDown] + U[IdxLeft + 1] -
                          2 * U[IdxLeft] + U[IdxLeft - 1]);
  }
  
  /************
   * j == 0
   ***********/
  if(j == 0 && i > 0 && i < SizeX - 1)
  {
    UNew[Idx] = -1 * UOld[IdxUp] + ABC_C1 * (UNew[IdxUp] + UNew[Idx]) +
                ABC_C2 * (U[Idx] + U[IdxUp]) +
                ABC_C3 * (U[IdxRight] - 2*U[Idx] + U[IdxLeft] +
                          U[IdxRight+1] - 2*U[IdxUp] + U[IdxLeft + 1]);

  }

  /**************
   *  j == SizeY
   *************/
  if(j == SizeY-1 && i > 0 && i < SizeX - 1)
  {
    UNew[Idx] = -1 * UOld[IdxDown] + ABC_C1 * (UNew[IdxDown] + UNew[Idx]) +
                ABC_C2 * (U[Idx] + U[IdxDown]) + 
                ABC_C3 * (U[IdxRight] - 2*U[Idx] + U[IdxLeft] + 
                          U[IdxRight-1] - U[IdxDown] + U[IdxLeft - 1]);
  }
  return;
}  // End of ApplyBC function

int main()
{

  /****************
   * Initialization
   ***************/
  int SizeX = 100;
  int SizeY = 100;
  int MaxTime = 1000; 

  int isrc = 50;
  int jsrc = 50;

  float dx = 0.001;
  float cc = 299792458.0;
  float dt = 0.99 / (sqrt(2) * cc);
  
  float caInit = dt*cc / dx;

  std::vector<float> h_OldU(SizeX*SizeY,0.0);
  std::vector<float> h_NewU(SizeX*SizeY,0.0);
  std::vector<float> h_U(SizeX*SizeY,0.0);
  std::vector<float> h_Ca(SizeX*SizeY,caInit*caInit);
  
  float* d_OldU;
  float* d_NewU;
  float* d_U;
  float* d_Ca;
  
  /******************
   *  Allocate Memory
   *****************/
  hipMalloc((void**)&d_OldU, h_OldU.size());
  hipMalloc((void**)&d_NewU, h_NewU.size());
  hipMalloc((void**)&d_U, h_U.size());
  hipMalloc((void**)&d_Ca, h_Ca.size());

  hipMemcpy(d_OldU, h_OldU.data(), h_OldU.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_NewU, h_NewU.data(), h_NewU.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_U, h_U.data(), h_U.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_Ca, h_Ca.data(), h_Ca.size(), hipMemcpyHostToDevice);
  
  int BlockSize = 32;
  int NumBlocks = (SizeX*SizeY - 1) / BlockSize + 1;

  for(int t = 0; t < MaxTime; t++)
  {
    Update<<<NumBlocks,BlockSize>>>(d_OldU,d_NewU,d_U,d_Ca,SizeX,SizeY,isrc,jsrc,t,dt);
    ApplyBC<<<NumBlocks,BlockSize>>>(d_OldU,d_NewU,d_U,SizeX,SizeY,dt,dx); 
    d_OldU = d_U;
    d_U = d_NewU;
  }

  /***************
   *  Free Memory
   **************/
  hipFree(d_OldU);
  hipFree(d_NewU);
  hipFree(d_U);
  hipFree(d_Ca);

  return 0;
}








